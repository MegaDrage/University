#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <fstream>

#define N 10000000  // Размер вектора

__global__ void vectorAdd(const float *A, const float *B, float *C, long n) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

float runKernel(long threadsPerBlock) {
    long blocksPerGrid = N / threadsPerBlock;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_A.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return milliseconds;
}

int main() {
    std::ofstream outFile("result.csv");
    // outFile << "Threads,Time (ms)\n";

    // std::cout << "Threads\tTime (ms)\n";
    for (long T = 2; T <= 1024; T += 2) {
        float time = runKernel(T);
        // std::cout << T << "\t" << time << " ms\n";
        outFile << T << " " << time << "\n";
    }

    outFile.close();
    // std::cout << "Results saved to result.csv\n";

    return 0;
}

